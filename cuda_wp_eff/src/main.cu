#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//https://proofwiki.org/wiki/Product_of_Triangular_Matrices

int max_per_row = 0;
__global__
void devTrianglesCount(int* col_indx,  int* csr_rows, int nnz, int rows, int* out_sum, int group_rows);

/** 
 * Description: Reads the data from the mtx files.
 * The first row contains 3 integers: rows columns of the sparse graph 
 * and the number of non zero elements. The non zero elements are stored in 
 * COO format. Also the data have one-based indexing. While reading them we tra them
 * to zero based indexing.
 * 
 * @param data char[] the name of the file to read
 * @param row_indx int*  where the rows of the nnz are stored
 * @param col_indx int*  where  the columns of the nnz are stored
 * @param nnz int* the number of non zero elements
 * @param rows int* the number of rows
 * @param cols itn* the number of columns
 */
void readData(char data[], int **row_indx, int **col_indx, int* nnz, int * rows, int* cols){
    FILE *f = fopen(data,"r");
    fscanf(f, "%d %d %d\n",rows, cols, nnz);
    printf("-READ %d %d %d\n",*rows,*cols,*nnz);
    col_indx[0] = (int*)malloc((*nnz)*sizeof(int));
    row_indx[0] = (int*)malloc((*nnz)*sizeof(int));
    for(int i = 0; i < *nnz; i++){
        fscanf(f, "%d %d", &col_indx[0][i] , &row_indx[0][i]);
        // data have 1 base index
        // transform to 0-based index
        col_indx[0][i]--;
        row_indx[0][i]--;
    }

    fclose(f);
}

/**
 * Description: Returns an array with the non zero rows in compressed format: (length rows insteadn of nnz).
 * Combined with the column index we have the CSR represantion of the sparse graph. Also finds the max non zero
 * elements per row and updates the global variable max_per_row
 * 
 * @param rows int 
 * @param nnz int
 * @param row_indx int* the row vector from the COO format.
 * 
 * 
 * Returns:
 *         csr_rows int*
 */
int* COOtoCSR(int rows, int nnz, int* row_indx){
    // initialize
    int* csr_rows = (int*)malloc(rows*sizeof(int));
    for(int i = 0; i < rows; i++){
        csr_rows[i] = 0;
    }

    // Transformation to CSR 
    for(int i = 0; i < nnz; i++){
        int index = row_indx[i]+1;
        if(index < rows)
            csr_rows[index]++;
    }
    for(int i = 1; i < rows; i++){
        if(csr_rows[i] > max_per_row){
            max_per_row = csr_rows[i];
        }
        csr_rows[i] += csr_rows[i-1];
    }

    return csr_rows;
}

void printTime(struct timeval start, struct timeval end, char* str){
    unsigned long ss,es,su,eu,s,u;
    ss  =start.tv_sec;
    su = start.tv_usec;
    es = end.tv_sec;
    eu = end.tv_usec;
    s = es - ss;
    if(eu > su){
        u = eu - su;
    }else{
        s--;
        u = 1000000 + eu - su;
    }
   
    printf("%s,%lu,%lu\n",str,s,u);
}






int main(int argc, char** argv){

    if(argc != 2){
        printf("Invalid arguments\n");
        return 1;
    }



    //cudaDeviceReset();
    struct timeval start,end,ALLSTART,ALLEND;

    // "auto.mtx"; // "data.csv"; //  "great-britain_osm.mtx"; // "delaunay_n22.mtx"; //
    printf("-Dataset: %s\n",argv[1]);
    int rows,cols,nnz;
    int *col_indx, *row_indx;
    int sum;

    /* Read Data in COO format and transform to 0 based index */
    gettimeofday(&start,NULL);
    readData(argv[1],&row_indx,&col_indx,&nnz,&rows,&cols);
    gettimeofday(&end,NULL);
    printTime(start,end, "Read Data");


        
    // Transform to CSR
    gettimeofday(&start,NULL);
    int* csr_rows = COOtoCSR(rows, nnz, row_indx);
    // We no longer need row_indx since we have csr_rows
    free(row_indx);
    gettimeofday(&end,NULL);
    printTime(start,end, "CSR");
    
    printf("-MAX PER ROW = %d\n",max_per_row);
    
    gettimeofday(&start,NULL);
    hipError_t cuer;
    int *cu_col_indx,  *cu_csr_rows;
    int* cu_sum;
    cuer = hipMalloc(&cu_col_indx,nnz*sizeof(int));
    printf("-%s\n",hipGetErrorName(cuer));
    cuer = hipMalloc(&cu_csr_rows,rows*sizeof(int));
    printf("-%s\n",hipGetErrorName(cuer));
    cuer = hipMalloc(&cu_sum,rows*sizeof(int));
    printf("-%s\n",hipGetErrorName(cuer));
    

    cuer = hipMemcpy(cu_col_indx,col_indx,nnz*sizeof(int),hipMemcpyHostToDevice);
    printf("-%s\n",hipGetErrorName(cuer));
    cuer = hipMemcpy(cu_csr_rows,csr_rows,rows*sizeof(int),hipMemcpyHostToDevice);
    printf("-%s\n",hipGetErrorName(cuer));

    int* res = (int*)malloc(rows*sizeof(int));
    for(int i = 0; i < rows; i++){
        res[i] = 0;
    }

    hipMemcpy(cu_sum,res,rows*sizeof(int),hipMemcpyHostToDevice);

    gettimeofday(&end,NULL);
    printTime(start,end, "CUDA data transfer");


    gettimeofday(&start,NULL);
    //rows = 100;
    int threads = max_per_row;
    if(max_per_row > 64){
        return 1;
    }

    int group_rows = 64/threads;
    if(group_rows > 8){
        group_rows = 8;
    }

    threads = threads * group_rows;

    int blocksize = (1 + rows/group_rows)/(512*512) + 1;
    printf("-blocksize %d %d\n", blocksize, 512*512);

    printf("Group number: %d\n",group_rows);
    printf("Threads = MaxNNZ*group_rows: %d %d %d \n",threads,max_per_row,group_rows);
    printf("Row span = %d * %d = %d | actual rows %d\n",blocksize*512*512, group_rows, blocksize*group_rows*512*512,rows);
    devTrianglesCount<<<dim3(512,512,blocksize),threads>>>(cu_col_indx, cu_csr_rows, nnz, rows, cu_sum, group_rows);
    printf("-%s\n",hipGetErrorName(cuer));




    cuer = hipMemcpy(res,cu_sum,rows*sizeof(int),hipMemcpyDeviceToHost);
    printf("-%s\n",hipGetErrorName(cuer));


    sum = 0;
    for(int i = 0; i < rows; i++){
        if(res[i] > 0)
            sum += res[i];
    }

    printf("-Cuda triangles = %d\n",sum);
    gettimeofday(&end,NULL);
    printTime(start,end,"CUDA");
    
}


__global__
void devTrianglesCount(int* col_indx, int* csr_rows, int nnz, int num_of_rows, int* out_sum, int group_rows){
    int row = blockIdx.x*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + blockIdx.z;
    int id = threadIdx.x;
    int own_group = -1;
    int group_offset = 0;
    if(row*group_rows >= num_of_rows){
        return;
    }
    //max group number = 16
    __shared__ int sh_group_rows;
    __shared__ int start_row[16];
    __shared__ int end_row[16];
    __shared__ int len[16];
    __shared__ int* row_ptr;
    __shared__ int current_row[64]; 

    __shared__ int sh_len[64];
    __shared__ int* sh_ptr[64];
    __shared__ int sh_cols[64][64];
    __shared__ int sh_sum[64];
    sh_sum[id] = 0;
    end_row[id] = 0;
    start_row[id] = 0;
    __syncthreads();
    // Get the current rows
    sh_len[id] = 0;
    if(id == 0){
        sh_group_rows = group_rows-1;
        for(int i = 0; i < group_rows; i++){
            int t_row = row*group_rows + i;  // temp row
            start_row[i] = csr_rows[t_row];
            if(t_row == num_of_rows - 1){
                sh_group_rows = i-1;
                end_row[i] = nnz;
            }else{
                end_row[i] = csr_rows[t_row+1];
            }
            len[i] = end_row[i] - start_row[i];
        }
        row_ptr = &col_indx[start_row[0]];
    }
    __syncthreads();
    
    // if(id == 0){
    //     start_row = csr_rows[row];
    //     if(row == num_of_rows-1){
    //         end_row = nnz;
    //     }else{
    //         end_row = csr_rows[row+1];
    //     }
    //     len = end_row - start_row;
    //     row_ptr = &col_indx[start_row];
    // }
    // __syncthreads();

    if(id < end_row[sh_group_rows] - start_row[0]){
        current_row[id] = row_ptr[id];
    }
    __syncthreads();

    // Assign each thread to a group

    for(int i = 0; i < sh_group_rows+1; i++){
        //printf("len %d \n",end_row[i] - start_row[0]);
        if(id < end_row[i] - start_row[0]){
            own_group = i;
            group_offset = 0;
            if(i > 0){
                group_offset = end_row[i-1]-start_row[0];
            }
            break;
        }
    }


    __syncthreads();
   if(row < 50){
      // printf("id %d group offset %d \n",id,group_offset);
   }
    // Get info for each column
    if(own_group >= 0){
        
        int tmp_col = current_row[id];
        //printf("ID %d, group %d %d, row %d, len %d , own %d END %d START %d\n", id,sh_group_rows, group_offset, row,len[0], own_group,tmp_col,1);
        int tmp_start = csr_rows[tmp_col];
        int tmp_end;
        if(tmp_col == num_of_rows-1){
            tmp_end = nnz;
        }else{
            tmp_end = csr_rows[tmp_col+1];
        }
        sh_len[id] =  tmp_end - tmp_start;
        sh_ptr[id] = &col_indx[tmp_start];
    }

    __syncthreads();
  
    for(int i = 0; i < end_row[sh_group_rows]-start_row[0]; i++){
        if(id < sh_len[i]){
            sh_cols[i][id] = sh_ptr[i][id];
        }
    }
    __syncthreads();

    if(own_group >= 0){
        int a = 0;
        int b = 0;
        int sum = 0;
        while(1){
            if(a == len[own_group] || b == sh_len[id]){
                break;
            }

            int b1 = current_row[a + group_offset] == sh_cols[id][b];
            int b2 = current_row[a + group_offset] > sh_cols[id][b];
            int b3 = current_row[a + group_offset] < sh_cols[id][b];

            a = a + b1 + b3;
            b = b + b1 + b2;
            sum = sum + b1;
        }

        sh_sum[id] = sum;
    }
    __syncthreads();
    if(id == 0){
        int sum = 0;
        for(int i = 0; i < end_row[sh_group_rows]-start_row[0]; i++){
            sum += sh_sum[i];
        }
        out_sum[row] = sum;
    }
    __syncthreads();
   
    
}
